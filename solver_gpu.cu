#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "constants.hpp"

__global__ void fluidKernel(glm::vec2 *d_positions, glm::vec2 *d_gradients, uint32_t *d_neighbours, uint32_t *d_neighbour_counts, const uint32_t n) {
	uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;

	float rho = 0.0f;		
	float sum_grad2 = 0.0f;
	glm::vec2 grad_i(0.0f);

	const uint32_t query_size = d_neighbour_counts[i];

	for (uint32_t j = 0; j < query_size; j++) {
		const uint32_t nb_i = d_neighbours[i * n + j];
		glm::vec2 n = d_positions[nb_i] - d_positions[i];
		const float r2 = glm::dot(n,n);
					
		if (r2 > constants::h2) {
			d_gradients[nb_i] = glm::vec2(0.0f);
		} else {
			// Normalise.
			const float r = sqrt(r2);
			if (r > 0.0f) n /= r;

			const float w = constants::h2 - r2;
			rho += constants::kernel_scale * w * w * w;
			const float grad = (constants::kernel_scale * 3.0f * w * w * (-2.0f * r)) / constants::rest_density;
			d_gradients[nb_i] = n * grad;
			grad_i -= n * grad;
			sum_grad2 += grad * grad;	
		}
	}		
	
	sum_grad2 += glm::dot(grad_i,grad_i);
	const float c = rho / constants::rest_density - 1.0f;	
	const float lambda = -c / (sum_grad2 + constants::eps);

	for (uint32_t j = 0; j < query_size; j++) {
		const uint32_t nb_i = d_neighbours[i * n + j];
		if (i == nb_i) {
			d_positions[nb_i] += lambda * grad_i;
		} else {
			d_positions[nb_i] += lambda * d_gradients[nb_i];
		}
	}
}

void solveFluidGpu(glm::vec2 *h_positions, glm::vec2 *h_gradients, uint32_t *h_neighbours, uint32_t *h_neighbour_counts, const uint32_t n) {
	// Allocate memory on device.	
	glm::vec2 *d_positions;
	glm::vec2 *d_gradients;
	uint32_t *d_neighbours; // This 2d array is flattened.
	uint32_t *d_neighbour_counts;

	hipMalloc((void**)&d_positions, n * sizeof(glm::vec2));
	hipMalloc((void**)&d_gradients, n * sizeof(glm::vec2));
	hipMalloc((void**)&d_neighbours, n * n * sizeof(uint32_t));
	hipMalloc((void**)&d_neighbour_counts, n * sizeof(uint32_t));
	
	// Copy data from host to device.	
	hipMemcpy(d_positions, h_positions, n * sizeof(glm::vec2), hipMemcpyHostToDevice);
	hipMemcpy(d_gradients, h_gradients, n * sizeof(glm::vec2), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbours, h_neighbours , n * n * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbours, h_neighbour_counts, n * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	uint32_t threads_per_block = 256;
	uint32_t blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
	
	fluidKernel<<<blocks_per_grid, threads_per_block>>>(d_positions, d_gradients, d_neighbours, d_neighbour_counts, n);

	// Copy results from device to host.
	hipMemcpy(h_positions, d_positions, n * sizeof(glm::vec2), hipMemcpyDeviceToHost);
	hipMemcpy(h_gradients, d_gradients, n * sizeof(glm::vec2), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbours, d_neighbours, n * n * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbour_counts, d_neighbour_counts, n * sizeof(uint32_t), hipMemcpyDeviceToHost);
	
	hipFree(d_positions);
	hipFree(d_gradients);
	hipFree(d_neighbours);
	hipFree(d_neighbour_counts);
}
